#include "hip/hip_runtime.h"
#include "pre_and_post_processor_cuda.cuh"
#define MAX_THREADS 1024

#include <iostream>

/******************************************************************************
 * gpu_reorient: re-orientation and/or re-ordering of the axes
 *
 * Arguments:
 *      data: input data
 *      data_o: output data
 *      cord0: current ordinal number of the first axis
 *      cord1: current ordinal number of the second axis
 *      cord2: current ordinal number of the third axis
 *      dord0: desired ordinal number of the first axis
 *      dord1: desired ordinal number of the seconds axis
 *      dord2: desired ordinal number of the third axis
 *      corient0: current orientation of the first axis
 *      corient1: current orientation of the seconds axis
 *      corient2: current orientation of the third axis
 *      dorient0: desired orientation of the first axis
 *      dorient1: desired orientation of the seconds axis
 *      dorient2: desired orientation of the third axis
 *
 *****************************************************************************/
template<typename T>
__global__ void gpu_reorient(T *data, T *data_o,
                             unsigned w, unsigned h, unsigned d,
                             unsigned cord0, unsigned cord1, unsigned cord2,
                             unsigned dord0, unsigned dord1, unsigned dord2,
                             short corient0, short corient1, short corient2,
                             short dorient0, short dorient1, short dorient2)
{

    unsigned int t = (cord0 == 0) * w + (cord1 == 0) * h + (cord2 == 0) * d;

    if(threadIdx.x < t)
    {
        unsigned int in_idx = blockIdx.y * gridDim.x * t +
                              blockIdx.x * t + threadIdx.x;
        unsigned int out_idx = 0;

        unsigned int crs[3] = {threadIdx.x, blockIdx.x, blockIdx.y};
        unsigned int dord[3] = {dord0, dord1, dord2};
        unsigned int s[3] = {w, h, d};
        unsigned int cord[3] = {cord0, cord1, cord2};

        out_idx += crs[cord[dord[0]]];

        if(corient1 == dorient1)
            out_idx += crs[cord[dord[1]]] * s[dord[0]];
        else
            out_idx += (s[dord[1]] - 1 - crs[cord[dord[1]]]) * s[dord[0]];

        if(corient2 == dorient2)
            out_idx += crs[dord[cord[2]]] * s[dord[0]] * s[dord[1]];
        else
            out_idx += (s[dord[2]] - 1 - crs[dord[cord[2]]]) *
                s[dord[0]] * s[dord[1]];

        data_o[out_idx] = data[in_idx];
    }
}

/******************************************************************************
 * gpu_normalize: clip and normalize input data
 *
 * Arguments:
 *      volume: input volume
 *      w: volume width
 *      h: volume height
 *      d: volume depth
 *      lower_th_: lower clip value
 *      upper_th_: upper clip value
 *      minimum_value_: minimum output value
 *      maximum_value_: maximum output value
 *
 *****************************************************************************/
__global__ void gpu_normalize(float *volume,
                              unsigned w, unsigned h, unsigned d,
                              float lower_th_, float upper_th_,
                              float minimum_value_, float maximum_value_)
{

    unsigned int idx = blockIdx.y * gridDim.x * blockDim.x +
                       blockIdx.x * blockDim.x + threadIdx.x;

    if(idx < w * h * d)
    {
        if (volume[idx] < lower_th_)
            volume[idx] = minimum_value_;
        else if (volume[idx] > upper_th_)
            volume[idx] = maximum_value_;
        else
            volume[idx] = (volume[idx] - lower_th_) *
                          (maximum_value_ - minimum_value_) /
                          (upper_th_ - lower_th_) + minimum_value_;
    }
}

/******************************************************************************
 * reorient_permute: determine if there is a need to re-orient and/or permute
 * axes
 *
 * Arguments:
 *      volume: input volume
 *      w: volume width
 *      h: volume height
 *      d: volume depth
 *      lower_th_: lower clip value
 *      upper_th_: upper clip value
 *      minimum_value_: minimum output value
 *      maximum_value_: maximum output value
 *
 *****************************************************************************/
void reorient_permute(bool &reorient, bool &permute,
                      unsigned *cord, short *cornt,
                      unsigned *dord, short *dornt)
{
    for(unsigned int i = 1; i < 3; i++)
    {
        if(cornt[i] != dornt[i])
            reorient = true;
        if(cord[i] != dord[i])
            permute = true;
    }
}

/******************************************************************************
 * preprocess_volume_cuda: normalize voxel intensities and re-orient volume
 * axes if necessary
 *
 * Arguments:
 * 		in_volume: volume to be processed
 * 		w: volume width
 * 		h: volume height
 * 		d: volume depth / number of slices
 *      cord - current order of the axes
 *      cornt - current orientation of the axes
 * 		lower_threshold: lower limit for voxel intensity
 * 		upper_threshold: upper limit for voxel intensity
 * 		minimum_value: minimum voxel intensity value in the
 * 			normalized voxel range
 * 		maximum_value: maximum voxel intensity value in the
 * 			normalized voxel range
 *
 *****************************************************************************/
void preprocess_volume_cuda(float *in_volume,
                            unsigned int w, unsigned int h, unsigned int d,
                            unsigned int *cord, short *cornt,
                            float lower_threshold, float upper_threshold,
                            float minimum_value, float maximum_value)
{
    short dornt[3] = {0, 1, 1};
    unsigned dord[3] = {0, 1, 2};

    bool reorient = false;
    bool permute = false;
    reorient_permute(reorient, permute, cord, cornt, dord, dornt);

    float *volume_d;
    unsigned int volume_B = h * w * d * sizeof(float);

    hipMalloc((void **) &volume_d, volume_B);
    hipMemcpy(volume_d, in_volume, volume_B, hipMemcpyHostToDevice);

    unsigned int i1, i2;
    i1 = (cord[1] == 0) * w + (cord[1] == 1) * h + (cord[1] == 2) * d;
    i2 = (cord[2] == 0) * w + (cord[2] == 1) * h + (cord[2] == 2) * d;

    dim3 grid(i1, i2);

    gpu_normalize<<<grid, MAX_THREADS>>>(volume_d,
                                         w, h, d,
                                         lower_threshold, upper_threshold,
                                         minimum_value, maximum_value);
    if(reorient or permute)
    {
        float *volume_o_d;
        hipMalloc((void **) &volume_o_d, volume_B);
        gpu_reorient<float><<<grid, MAX_THREADS>>>
                (volume_d, volume_o_d, w, h, d,
                 cord[0], cord[1], cord[2], dord[0], dord[1], dord[2],
                 cornt[0], cornt[1], cornt[2], dornt[0], dornt[1], dornt[2]);
        hipMemcpy(in_volume, volume_o_d, volume_B, hipMemcpyDeviceToHost);
        hipFree(volume_o_d);
    }
    else
        hipMemcpy(in_volume, volume_d, volume_B, hipMemcpyDeviceToHost);
    hipFree(volume_d);
}

/******************************************************************************
 * normalize_volume_cuda: normalize voxel intensities
 *
 * Arguments:
 *      in_volume: volume to be processed
 *      w: volume width
 *      h: volume height
 *      d: volume depth / number of slices
 *      lower_threshold: lower limit for voxel intensity
 *      upper_threshold: upper limit for voxel intensity
 *      minimum_value: minimum voxel intensity value in the
 *          normalized voxel range
 *      maximum_value: maximum voxel intensity value in the
 *          normalized voxel range
 *
 *****************************************************************************/
void normalize_volume_cuda(float *in_volume,
                           unsigned int w, unsigned int h, unsigned int d,
                           float lower_threshold, float upper_threshold,
                           float minimum_value, float maximum_value)
{

    float *volume_d;
    unsigned int volume_B = h * w * d * sizeof(float);

    hipMalloc((void **) &volume_d, volume_B);
    hipMemcpy(volume_d, in_volume, volume_B, hipMemcpyHostToDevice);

    dim3 grid(h, d);

    gpu_normalize<<<grid, MAX_THREADS>>>(volume_d, w, h, d,
                                         lower_threshold, upper_threshold,
                                         minimum_value, maximum_value);
    hipMemcpy(in_volume, volume_d, volume_B, hipMemcpyDeviceToHost);
    hipFree(volume_d);
}

/******************************************************************************
 * reorient_volume_cuda: re-orient axes of volume if necessary
 *
 * Arguments:
 *      in_volume: volume to be reoriented
 *      w: volume width
 *      h: volume height
 *      d: volume depth / number of slices
 *      cord - current order of the axes
 *      cornt - current orientation of the axes
 *      dord - desired order of the axes
 *      dornt - desired orientation of the axes
 *
 *****************************************************************************/
void reorient_volume_cuda(float *in_volume,
                          unsigned int w, unsigned int h, unsigned int d,
                          unsigned *cord, short *cornt,
                          unsigned *dord, short *dornt)
{
    bool reorient = false;
    bool permute = false;
    reorient_permute(reorient, permute, cord, cornt, dord, dornt);

    if(reorient or permute)
    {
        float *volume_d;
        float *volume_o_d;
        unsigned int volume_B = h * w * d * sizeof(float);

        hipMalloc((void **) &volume_d, volume_B);
        hipMemcpy(volume_d, in_volume, volume_B, hipMemcpyHostToDevice);
        hipMalloc((void **) &volume_o_d, volume_B);

        unsigned int i1, i2;
        i1 = (cord[1] == 0) * w + (cord[1] == 1) * h + (cord[1] == 2) * d;
        i2 = (cord[2] == 0) * w + (cord[2] == 1) * h + (cord[2] == 2) * d;

        dim3 grid(i1, i2);

        gpu_reorient<float><<<grid, MAX_THREADS>>>
                (volume_d, volume_o_d, w, h, d,
                 cord[0], cord[1], cord[2], dord[0], dord[1], dord[2],
                 cornt[0], cornt[1], cornt[2], cornt[0], cornt[1], cornt[2]);

        hipMemcpy(in_volume, volume_d, volume_B, hipMemcpyDeviceToHost);
        hipFree(volume_d);
        hipFree(volume_o_d);
    }
}

/******************************************************************************
 * reorient_segmentation_cuda: re-orient axes of segmentation if necessary
 *
 * Arguments:
 *      in_segment: segmentation to be reoriented
 *      w: volume width
 *      h: volume height
 *      d: volume depth / number of slices
 *      cord - current order of the axes
 *      cornt - current orientation of the axes
 *      dord - desired order of the axes
 *      dornt - desired orientation of the axes
 *
 *****************************************************************************/
void reorient_segment_cuda(unsigned char *in_segment,
                           unsigned int w, unsigned int h, unsigned int d,
                           unsigned *cord, short *cornt,
                           unsigned *dord, short *dornt)
{
    bool reorient = false;
    bool permute = false;
    reorient_permute(reorient, permute, cord, cornt, dord, dornt);

    if(reorient or permute)
    {
        unsigned char *segment_d;
        unsigned char *segment_o_d;
        unsigned int segment_B = h * w * d * sizeof(unsigned char);

        hipMalloc((void **) &segment_d, segment_B);
        hipMemcpy(segment_d, in_segment, segment_B, hipMemcpyHostToDevice);
        hipMalloc((void **) &segment_o_d, segment_B);

        unsigned int i1, i2;
        i1 = (cord[1] == 0) * w + (cord[1] == 1) * h + (cord[1] == 2) * d;
        i2 = (cord[2] == 0) * w + (cord[2] == 1) * h + (cord[2] == 2) * d;

        dim3 grid(i1, i2);
        gpu_reorient<unsigned char><<<grid, MAX_THREADS>>>
                (segment_d, segment_o_d, w, h, d,
                 cord[0], cord[1], cord[2], dord[0], dord[1], dord[2],
                 cornt[0], cornt[1], cornt[2], dornt[0], dornt[1], dornt[2]);

        hipMemcpy(in_segment, segment_o_d, segment_B, hipMemcpyDeviceToHost);
        hipFree(segment_d);
        hipFree(segment_o_d);
    }
}
