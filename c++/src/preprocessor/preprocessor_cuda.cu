#include "hip/hip_runtime.h"
#include "preprocessor_cuda.h"


__global__ void gpu_normalization(float *cuda_volume, bool change_direction,
		                          float lower_threshold_, float upper_threshold_,
		                          float minimum_value_, float maximum_value_)
{
	unsigned int in_idx = blockIdx.y * gridDim.x * blockDim.x +
			              blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int out_idx;

	if(change_direction)
		out_idx = blockIdx.y * gridDim.x * blockDim.x +
                  (gridDim.x - blockIdx.x) * blockDim.x + threadIdx.x;
	else
		out_idx = in_idx;

	if(cuda_volume[out_idx] < lower_threshold_)
		cuda_volume[in_idx] = maximum_value_;
	else if(cuda_volume[in_idx] > upper_threshold_)
		cuda_volume[out_idx] = maximum_value_;
	else
		cuda_volume[out_idx] =
				(cuda_volume[in_idx] - upper_threshold_) *
				(maximum_value_ - minimum_value_) /
				(lower_threshold_ - upper_threshold_) + minimum_value_;
}

void preprocess_cuda(float *input_volume,
		             unsigned int h, unsigned int w, unsigned int d,
		             bool change_direction,
		             float lower_threshold, float upper_threshold,
		             float minimum_value, float maximum_value)
{
	float *cuda_volume;
	unsigned int volume_bytes = h * w * d * sizeof(float);

	hipMalloc((void **)&cuda_volume, volume_bytes);
	hipMemcpy(cuda_volume, input_volume, volume_bytes, hipMemcpyHostToDevice);

	dim3 grid(h, d);

	gpu_normalization<<<grid, w>>>(cuda_volume, change_direction,
			                       lower_threshold, upper_threshold,
			                       minimum_value, maximum_value);

	hipMemcpy(input_volume, cuda_volume, volume_bytes, hipMemcpyDeviceToHost);
	hipFree(cuda_volume);
}

